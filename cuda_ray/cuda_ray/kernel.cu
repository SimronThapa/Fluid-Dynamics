#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <string>
#include <fstream>
#include <sstream>
#include <iterator>
#include <stdlib.h>
#include <time.h>
#include <float.h>

#include <hiprand/hiprand_kernel.h>

#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "triangle.h"
#include "hitable_list.h"
#include "camera.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

using namespace std;
struct VERTEX
{
	float x, y, z;
};
struct FACE
{
	int x, y, z;
};

void ReadOBJFile(const char filename[], float **vertex, int **face, int& VN, int& FN) {
	vector<VERTEX> Vertex;
	vector<FACE> Face;
	std::ifstream file(filename);
	thrust::host_vector<std::string> fileContents;

	//check file existance
	if (!file)
	{
		std::cerr << "Failure opening file at \"" << filename << "\".";
	}

	//file is located
	std::string buffer;
	while (std::getline(file, buffer))
	{
		// Add the buffer contents to our fileContents vector if it's not a comment
		// (Doing the check now reduces memory usage
		if (buffer[0] != '#' || buffer[0] != ' ')
		{
			fileContents.push_back(buffer);
		}
	}
	if (fileContents.size() == 0)
	{
		std::cerr << "File \"" << filename << "\" Was empty... Failure to load\n";
	}
	//save the vertices and faces in the structure
	for (unsigned int n = 0; n < fileContents.size(); n++) {
		//std::cout << fileContents[n].c_str()[0] << "\n";
		
		if (fileContents[n].c_str()[0] == 'v')
		{
			float tmpx, tmpy, tmpz;
			sscanf(fileContents[n].c_str(), "v %f %f %f", &tmpx, &tmpy, &tmpz);
			VERTEX tmpVert = { tmpx, tmpy, tmpz };
			//cout <<"v " <<tmpVert.x << " " << tmpVert.y << " " << tmpVert.z <<"\n";
			Vertex.push_back(tmpVert);
		}
		else if (fileContents[n].c_str()[0] == 'f') {
			int tmpx, tmpy, tmpz;
			sscanf(fileContents[n].c_str(), "f %d %d %d", &tmpx, &tmpy, &tmpz);
			FACE tmpFace = { tmpx, tmpy, tmpz };
			//cout << "f " << tmpFace.x <<" "<< tmpFace.y << " " << tmpFace.z << "\n";
			Face.push_back(tmpFace);
		}
	}

	if ((Vertex.size() != 0) || (Face.size() != 0))
	{
		VN = Vertex.size();
		FN = Face.size();
		cout << "This .obj file has " << Vertex.size() << " vertexs" << endl;
		cout << "This .obj file has " << Face.size() << " faces" << endl;
	}
	for (int i = 0; i < 3; ++i) {
		vertex[i] = new float[Vertex.size()];
		face[i] = new int[Face.size()];
	}
	for (int i = 0; i < Vertex.size(); i++)
	{
		vertex[0][i] = Vertex[i].x;
		vertex[1][i] = Vertex[i].y;
		vertex[2][i] = Vertex[i].z;
	}
	for (int i = 0; i < Face.size(); i++)
	{
		face[0][i] = Face[i].x;
		face[1][i] = Face[i].y;
		face[2][i] = Face[i].z;
	}

}
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
	if (result) {
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
			file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		exit(99);
	}
}

__device__ vec3 color(const ray& r, hitable **world) {
	hit_record rec;
	if ((*world)->hit(r, 0.0, FLT_MAX, rec)) {
		printf("it hit!\n");
		return 0.5f*vec3(rec.normal.x() + 1.0f, rec.normal.y() + 1.0f, rec.normal.z() + 1.0f);
	}
	else {
		printf("it not hit!\n");
		vec3 unit_direction = unit_vector(r.direction());
		float t = 0.5f*(unit_direction.y() + 1.0f);
		return (1.0f - t)*vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
	}
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j * max_x + i;
	//Each thread gets same seed, a different sequence number, no offset
	hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3 *fb, int max_x, int max_y, int ns, camera **cam, hitable **world /*,hiprandState *rand_state*/) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j * max_x + i;
	/*
	hiprandState local_rand_state = rand_state[pixel_index];
	vec3 col(0, 0, 0);
	for (int s = 0; s < ns; s++) {
		float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
		float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
		ray r = (*cam)->get_ray(u, v);
		col += color(r, world);
	}
	printf("color = %f \n", col / float(ns));
	fb[pixel_index] = col / float(ns);
	*/

	float u = float(i) / float(max_x);
	float v = float(j) / float(max_y);
	ray r = (*cam)->get_ray(u, v);
	fb[pixel_index] = color(r, world);

}

__global__ void create_world(hitable **d_list, hitable **d_world, camera **d_camera, float *vertex,  int *face, int VN, int FN) {
	int index = threadIdx.x;
	int stride = blockDim.x;		
	for (int i = index; i < FN * 3; i+= stride) {
		int idx1 = face[i]; i++;
		int idx2 = face[i]; i++;
		int idx3 = face[i];
		if(i == index)
			*(d_list) = new triangle(vec3(vertex[idx1 * 3], vertex[idx1 * 3 + 1], vertex[idx1 * 3 + 2]), vec3(vertex[idx2 * 3 ], vertex[idx2 * 3 + 1], vertex[idx2 * 3 + 2]), vec3(vertex[idx3 * 3], vertex[idx3 * 3 + 1], vertex[idx3 * 3 + 2]), vec3(0, 0, 1), 1);
		else
			*(d_list + i) = new triangle(vec3(vertex[idx1 * 3], vertex[idx1 * 3 + 1], vertex[idx1 * 3 + 2]), vec3(vertex[idx2 * 3], vertex[idx2 * 3 + 1], vertex[idx2 * 3 + 2]), vec3(vertex[idx3 * 3], vertex[idx3 * 3 + 1], vertex[idx3 * 3 + 2]), vec3(0, 0, 1), 1);
		
		//printf("%f, %f,%f \n", vertex[idx1 * 3], vertex[idx1 * 3 + 1], vertex[idx1 * 3 + 2]);
		//printf("%f, %f,%f \n", vertex[idx2 * 3], vertex[idx2 * 3 + 1], vertex[idx2 * 3 + 2]);
		//printf("%f, %f,%f \n", vertex[idx3 * 3], vertex[idx3 * 3 + 1], vertex[idx3 * 3 + 2]);
	}
	//*(d_list + 1) = new sphere(vec3(0, 0, -1), 0.5);
	*d_world = new hitable_list(d_list, sizeof(face));
	*d_camera = new camera();

}

__global__ void free_world(hitable **d_list, hitable **d_world, camera **d_camera, int FN) {
	delete *(d_list);
	//delete *(d_list + 1);
	for (int i = 1; i < FN; i++) {
		delete *(d_list + i );
	}
	delete *d_world;
	delete *d_camera;
}

int main() {
	int nx = 512;
	int ny = 512;
	int ns = 100;
	int tx = 8;
	int ty = 8;

	std::cerr << "Rendering a " << nx << "x" << ny << " image with " << ns << " samples per pixel ";
	std::cerr << "in " << tx << "x" << ty << " blocks.\n";

	int num_pixels = nx * ny;
	size_t fb_size = num_pixels * sizeof(vec3);

	// allocate FB
	vec3 *fb;
	checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

	// allocate random state
	hiprandState *d_rand_state;
	checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels * sizeof(hiprandState)));

	// make our world of hitables & the camera
	hitable **d_list;
	checkCudaErrors(hipMalloc((void **)&d_list, 2 * sizeof(hitable *)));
	hitable **d_world;
	checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable *)));
	camera **d_camera;
	checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));

	// copy the vertex and face from host to device 
	float **VertexK = new float *[3];
	int **FaceK = new int*[3];
	int VN = 0, FN = 0;
	ReadOBJFile("G:\\outputFile\\data_part77.obj", VertexK, FaceK, VN, FN);
	
	// Allocate Unified Memory � accessible from CPU or GPU
	float *Vertex;
	hipMallocManaged(&Vertex, sizeof(float)* VN * 3);

	int *Face;
	hipMallocManaged(&Face, sizeof(int)* FN * 3);

	//unified variable initialization
	int counter = 0;
	for (int i = 0; i < FN; i++) {
		Face[counter] = FaceK[0][i]; counter++;
		Face[counter] = FaceK[1][i]; counter++;
		Face[counter] = FaceK[2][i]; counter++;	
	}
	counter = 0;
	for (int i = 0; i < VN; i++) {
		Vertex[counter] = VertexK[0][i]; counter++;
		Vertex[counter] = VertexK[1][i]; counter++;
		Vertex[counter] = VertexK[2][i]; counter++;
	}

	// Run kernel on FN*3 elements on the GPU
	/*
		counter = 0;
		for (int i = 1; i <= VN*3; i++) {
			printf("%f ", Vertex[i-1]);
			if (i % 3 == 0)
				printf("\n");
		}
	*/
	
	create_world << <1, 1 >> > (d_list, d_world, d_camera, Vertex, Face, VN, FN);
	
	// Wait for GPU to finish before accessing on host
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	clock_t start, stop;
	start = clock();
	// Render our buffer
	dim3 blocks(nx / tx + 1, ny / ty + 1);
	dim3 threads(tx, ty);
	//render_init << <blocks, threads >> > (nx, ny, d_rand_state);
	//checkCudaErrors(hipGetLastError());
	//checkCudaErrors(hipDeviceSynchronize());
	render << <blocks, threads >> > (fb, nx, ny, ns, d_camera, d_world);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	stop = clock();
	double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
	std::cerr << "took " << timer_seconds << " seconds.\n";

	// Output FB as Image
	std::cout << "P3\n" << nx << " " << ny << "\n255\n";
	FILE *fp = fopen("test.ppm", "wb");
	fprintf(fp, "P6\n%d %d\n255\n", nx, ny);
	for (int j = ny - 1; j >= 0; j--) {
		for (int i = 0; i < nx; i++) {
			size_t pixel_index = j * nx + i;
			int ir = int(255.99*fb[pixel_index].r());
			int ig = int(255.99*fb[pixel_index].g());
			int ib = int(255.99*fb[pixel_index].b());
			//std::cout << ir << " " << ig << " " << ib << "\n";
			static unsigned char color[3];
			color[0] = ir;//r
			color[1] = ig;//g
			color[2] = ib;//b
			fwrite(color, 1, 3, fp);
		}
	}

	// clean up
	checkCudaErrors(hipDeviceSynchronize());
	free_world << <1, 1 >> > (d_list, d_world, d_camera, FN);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipFree(d_camera));
	checkCudaErrors(hipFree(d_world));
	checkCudaErrors(hipFree(d_list));
	checkCudaErrors(hipFree(d_rand_state));
	checkCudaErrors(hipFree(fb));
	checkCudaErrors(hipFree(Vertex));
	checkCudaErrors(hipFree(Face));
	

	// useful for cuda-memcheck --leak-check full
	hipDeviceReset();
}



